
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 2048
#define TILE_SIZE 32
#define M 16

__global__ void matMult_largeBlocks(const float *A, const float *B, float *C, int n)
{	
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	
	__shared__ float As[TILE_SIZE][TILE_SIZE];
	__shared__ float Bs[TILE_SIZE][TILE_SIZE];

	float Csub1 = 0.f;
	float Csub2 = 0.f;

	for (int b = 0; b < n/TILE_SIZE; ++b) {
		As[ty][tx] = A[(by * blockDim.y + ty) * n + (b * TILE_SIZE + tx)];
		As[ty+M][tx] = A[(by * blockDim.y + ty + M) * n + (b * TILE_SIZE + tx)];
		Bs[ty][tx] = B[(b * TILE_SIZE + ty) * n + (bx * blockDim.x + tx)];
		Bs[ty+M][tx] = B[(b * TILE_SIZE + ty + M) * n + (bx * blockDim.x + tx)];
		__syncthreads();

		#pragma unroll 32
		for (int k = 0; k < TILE_SIZE; ++k) {
			Csub1 += As[ty][k] * Bs[k][tx];
			Csub2 += As[ty+M][k] * Bs[k][tx];
		}
		__syncthreads();
	}

	C[(by * blockDim.y + ty) * n + (bx * blockDim.x + tx)] = Csub1;
	C[(by * blockDim.y + ty + M) * n + (bx * blockDim.x + tx)] = Csub2;
}

__global__ void matMult_tiled(const float *A, const float *B, float *C, int n)
{
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	__shared__ float As[TILE_SIZE][TILE_SIZE];
	__shared__ float Bs[TILE_SIZE][TILE_SIZE];

	float Csub = 0.f;
	for (int b = 0; b < n/TILE_SIZE; b++) {
		As[ty][tx] = A[(by * blockDim.y + ty) * n + (b * TILE_SIZE + tx)];
		Bs[ty][tx] = B[(b * TILE_SIZE + ty) * n + (bx * blockDim.x + tx)];
		__syncthreads();

		for (int k = 0; k < TILE_SIZE; k++) {
			Csub += As[ty][k] * Bs[k][tx];
		}
		__syncthreads();
	}

	C[(by * blockDim.y + ty) * n + (bx * blockDim.x + tx)] = Csub;
}

__global__ void matMult_naive(const float *A, const float *B, float *C, int n)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	if (row >= n || col >= n)
		return;

	float tmp = 0.f;
	int i;
	for (i = 0; i < n; ++i)
		tmp += A[row * n + i] * B[i * n + col];

	C[row * n + col] = tmp;
}

//void printMatrix(const char *name, float *M, int n)
//{
//	printf("%s:\n", name);
//
//	int i, j;
//	for (i = 0; i < n; ++i) {
//		for (j = 0; j < n; ++j) {
//			printf("%8.2f ", M[i * n + j]);
//		}
//		printf("\n");
//	}
//}

void fillMatrices(float *A, float *B)
{	
	for (int i = 0; i < N; ++i) {
		for (int j = 0; j < N; ++j) {
			A[i * N + j] = 10.f * (float) rand() / (float) RAND_MAX;
			B[i * N + j] = 10.f * (float) rand() / (float) RAND_MAX;
		}
	}
}

int main(void)
{

#if defined LARGE_BLOCKS
	size_t blockSize = TILE_SIZE;
	dim3 threadsPerBlock(blockSize, M);
	dim3 blocksPerGrid(ceil(N/blockSize), ceil(N/blockSize));
#elif defined TILED
	size_t blockSize = TILE_SIZE;
	dim3 threadsPerBlock(blockSize, blockSize);
	dim3 blocksPerGrid(ceil(N/blockSize), ceil(N/blockSize));
#else // NAIVE
	dim3 threadsPerBlock(16, 16);
	dim3 blocksPerGrid(ceil(N/16), ceil(N/16));
#endif


	int device = 0;
	if (hipSetDevice(device) != hipSuccess) {
		fprintf(stderr, "Could not set CUDA device!\n");
		return EXIT_FAILURE;
	}

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, device);
	printf("Using device %d: \"%s\"\n", device, deviceProp.name);	

	// setup timers
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float milliseconds = 0;

	size_t matrixSizeBytes = N * N * sizeof(float);
	int status = EXIT_FAILURE;
	float *hA, *hB, *hC;
	float *dA, *dB, *dC;
	hA = (float *) malloc(matrixSizeBytes);
	hB = (float *) malloc(matrixSizeBytes);
	hC = (float *) malloc(matrixSizeBytes);

	hipMalloc(&dA, matrixSizeBytes);
	hipMalloc(&dB, matrixSizeBytes);
	hipMalloc(&dC, matrixSizeBytes);

	if (!hA || !hB || !hC || !dA || !dB || !dC) {
		fprintf(stderr, "Could not allocate memory!\n");
		goto cleanup;
	}
	
	fillMatrices(hA, hB);

	hipMemcpy(dA, hA, matrixSizeBytes, hipMemcpyHostToDevice);
	hipMemcpy(dB, hB, matrixSizeBytes, hipMemcpyHostToDevice);
	hipMemcpy(dC, hC, matrixSizeBytes, hipMemcpyHostToDevice);

	hipEventRecord(start);
#if defined LARGE_BLOCKS
	matMult_largeBlocks<<<blocksPerGrid, threadsPerBlock>>>(dA, dB, dC, N);
#elif defined TILED
	matMult_tiled<<<blocksPerGrid, threadsPerBlock>>>(dA, dB, dC, N);
#else
	matMult_naive<<<blocksPerGrid, threadsPerBlock>>>(dA, dB, dC, N);
#endif

	hipEventRecord(stop);

	hipMemcpy(hC, dC, matrixSizeBytes, hipMemcpyDeviceToHost);

	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);

#if 0
	printMatrix("hA", hA, N);
	puts("+");
	printMatrix("hB", hB, N);
	puts("=");
	printMatrix("hC", hC, N);
#endif

#if defined LARGE_BLOCKS
	printf("Large blocks version\n");
#elif defined TILED
	printf("Tiled version\n");
#else
	printf("Naive version\n");	
#endif
	printf("N = %d\n", N);
	printf("Calculation status: %s\n", hC[0] != 0 ? "success" : "failed");
	printf("Threads per block: %u x %u = %u\n", threadsPerBlock.x, threadsPerBlock.y, threadsPerBlock.x * threadsPerBlock.y);
	printf("Blocks per grid: %u x %u = %u\n", blocksPerGrid.x, blocksPerGrid.y, blocksPerGrid.x * blocksPerGrid.y);
	printf("Elapsed time: %f ms\n", milliseconds);
	printf("GPU performance: %f megaevals/s\n", float(N*N)/milliseconds/1000.f);

	status = EXIT_SUCCESS;

cleanup:
	hipFree(dA); hipFree(dB); hipFree(dC);
	free(hA); free(hB); free(hC);

	return status;
}
