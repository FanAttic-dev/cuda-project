
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 2048
#define BLOCK_SIZE 32

__global__ void vecsum(float *V, int n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	V[i] = V[i] + V[i + blockDim.x];
}

void fillVector(float *v)
{	
	float sum = 0.f;
	for (int i = 0; i < N; ++i) {
		sum += i;
	}
	for (int i = 0; i < N; ++i) {
		v[i] = (float) i / sum;
	}
}

int main(void)
{
	int device = 0;
	if (hipSetDevice(device) != hipSuccess) {
		fprintf(stderr, "Could not set CUDA device!\n");
		return EXIT_FAILURE;
	}

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, device);
	printf("Using device %d: \"%s\"\n", device, deviceProp.name);	

	// setup timers
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float milliseconds = 0;

	int status = EXIT_FAILURE;
	float *hV;
	float *dV;
	size_t vecSizeBytes = N * sizeof(float);

	hV = (float *) malloc(vecSizeBytes);
	hipMalloc(&dV, vecSizeBytes);

	if (!hV || !dV) {
		fprintf(stderr, "Could not allocate memory!\n");
		goto cleanup;
	}
	
	fillVector(hV);

	hipMemcpy(dV, hV, vecSizeBytes, hipMemcpyHostToDevice);

	hipEventRecord(start);

	for (int n = N/2; n > 0; n /= 2) {
		vecsum<<<1, n>>>(dV, n);
	}

	hipEventRecord(stop);

	hipMemcpy(hV, dV, vecSizeBytes, hipMemcpyDeviceToHost);

	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);

	printf("N = %d\n", N);
	printf("Sum: %f\n", hV[0]);
	printf("Elapsed time: %f ms\n", milliseconds);
	printf("GPU performance: %f megaevals/s\n", float(N*N)/milliseconds/1000.f);

	status = EXIT_SUCCESS;

cleanup:
	hipFree(dV);
	free(hV);

	return status;
}
