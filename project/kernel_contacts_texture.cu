#include "hip/hip_runtime.h"
#define BLOCK_WIDTH 16
#define BLOCK_HEIGHT 16
#define REDUCTION_BLOCK_SIZE 512

texture<int, 2, hipReadModeElementType> tex_contacts;

__device__ void warp_reduce(volatile int *shared_data, int tid)
{
	shared_data[tid] += shared_data[tid + 32];
	shared_data[tid] += shared_data[tid + 16];
	shared_data[tid] += shared_data[tid + 8];
	shared_data[tid] += shared_data[tid + 4];
	shared_data[tid] += shared_data[tid + 2];
	shared_data[tid] += shared_data[tid + 1];
}

__device__ bool check_neighbours(const int* const city, const int n, const int x, const int y, const int threshold)
{
	// check neighbours
	int inf_neighbours = 0;
	for (int dy = -1; dy <= 1; ++dy) {
		for (int dx = -1; dx <= 1; ++dx) { 
			if (city[(y + dy) * n + (x + dx)] > 0)
				++inf_neighbours;
		}
	}

	return inf_neighbours > threshold;
}

__global__ void make_iteration(const int* const contacts, const int* const in, int* const out, const int n, const int iter, int* const iter_block_infections)
{
	__shared__ int shared_iter_block_infections[BLOCK_HEIGHT * BLOCK_WIDTH];
	const int idx_local = threadIdx.y * blockDim.x + threadIdx.x;
	shared_iter_block_infections[idx_local] = 0;

	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int x = blockIdx.x * blockDim.x + threadIdx.x;

	if (y >= n || x >= n)
		return;

	const int idx_global = y * n + x;

	const int house_in = in[idx_global];
	int house_out = 0;

	if (house_in == 0) { // healthy
		const int threshold = tex2D(tex_contacts, x, y);
		bool infected = check_neighbours(in, n, x, y, threshold);

		// compare to connectivity
		if (infected) {
			house_out = 10;
			++shared_iter_block_infections[idx_local];
		}
	} else if (house_in > 0) { // infected
		house_out = house_in - 1 == 0 ? -30 : house_in - 1;
	} else { // (house_in < 0) // recovering, immune
		house_out = house_in + 1;
	} 
	__syncthreads();

/*	
	// sum and save the total number of new infections in this iteration per block
	if (idx_local == 0) {
		int iter_block_idx = (iter * gridDim.x * gridDim.y) + (blockIdx.y * gridDim.x + blockIdx.x);
		iter_block_infections[iter_block_idx] = 0;
		for (int yy = 0; yy < BLOCK_SIZE; ++yy)
		for (int xx = 0; xx < BLOCK_SIZE; ++xx)
			iter_block_infections[iter_block_idx] += shared_iter_block_infections[yy * BLOCK_SIZE + xx];
	}
*/

	// reduction
	// sum and save the total number of new infections in this iteration per block

	for (unsigned int s = BLOCK_HEIGHT * BLOCK_WIDTH / 2; s > 32; s >>= 1) {
		if (idx_local < s) {
			shared_iter_block_infections[idx_local] += shared_iter_block_infections[idx_local + s];
		}
		__syncthreads();
	}
/*
	// TODO make generic
	if (idx_local < 128) {
		shared_iter_block_infections[idx_local] += shared_iter_block_infections[idx_local + 128];
		__syncthreads();
	}

	if (idx_local < 64) {
		shared_iter_block_infections[idx_local] += shared_iter_block_infections[idx_local + 64];
		__syncthreads();
	}
*/

	if (idx_local < 32) {
		warp_reduce(shared_iter_block_infections, idx_local);
		__syncthreads();
	}

	if (idx_local == 0) {
		int iter_block_idx = (iter * gridDim.x * gridDim.y) + (blockIdx.y * gridDim.x + blockIdx.x);
		iter_block_infections[iter_block_idx] = shared_iter_block_infections[0];
	}

	out[idx_global] = house_out;
}

/*
   For each iteration, sums infections per block to compute the number of new infections per iteration.
*/
__global__ void reduce_infections(int* const infections, const int* const iter_block_infections, const int iters, const int blocks_per_iter, const dim3 grid_size)
{
	__shared__ int shared[REDUCTION_BLOCK_SIZE];

	const int tid = threadIdx.x;
	const int iter = blockIdx.x;
	const int infections_idx = iter * grid_size.x * grid_size.y;
	shared[tid] = iter_block_infections[infections_idx + tid];

	for (int i = 1; i < blocks_per_iter; ++i) {
		shared[tid] += iter_block_infections[infections_idx + tid + (i * blockDim.x)];
	}

	__syncthreads();

	for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
		if (tid < s)
			shared[tid] += shared[tid + s];

		__syncthreads();
	}

	if (tid < 32) {
		warp_reduce(shared, tid);
		__syncthreads();
	}

	if (tid == 0) {
		infections[iter] = shared[0];
	}
}

void solveGPU(const int* const contacts, int* const city, int* const infections, const int n, const int iters)
{
	size_t size = n * n * sizeof(int);
	dim3 threads_per_block = dim3(BLOCK_WIDTH, BLOCK_HEIGHT);
	dim3 grid_size = dim3(ceil(n / (float) threads_per_block.x), ceil(n / (float) threads_per_block.y));

	int *in = city;
	int *out;
	int *iter_block_infections; 	// [iters][grid_size][grid_size] 
					// 3D array storing infections per block per iteration
	hipArray *tex_array_contacts;
	hipChannelFormatDesc channel_desc = 
		hipCreateChannelDesc(sizeof(int) * 8, 0, 0, 0, hipChannelFormatKindSigned);
	if ((hipMalloc((void**)&out, size) != hipSuccess)
			|| (hipMallocArray(&tex_array_contacts, &channel_desc, n, n) != hipSuccess)
		       	|| (hipMalloc((void**)&iter_block_infections, iters * grid_size.x * grid_size.y * sizeof(int)) != hipSuccess)) {
		fprintf(stderr, "hipMalloc failed\n");
		return;
	}

	tex_contacts.addressMode[0] = hipAddressModeBorder;
	tex_contacts.addressMode[1] = hipAddressModeBorder;
	tex_contacts.filterMode = hipFilterModePoint;
	tex_contacts.normalized = false;

	if (hipMemcpyToArray(tex_array_contacts, 0, 0, contacts, size, hipMemcpyDeviceToDevice) != hipSuccess) {
		fprintf(stderr, "hipMemcpyToArray failed\n");
		return;
	}

	if (hipBindTextureToArray(tex_contacts, tex_array_contacts, channel_desc) != hipSuccess) {
		fprintf(stderr, "hipBindTextureToArray failed\n");
		return;
	}

	dim3 blocks_per_grid = grid_size;
	for (int iter = 0; iter < iters; ++iter) {
		make_iteration<<<blocks_per_grid, threads_per_block>>>(contacts, in, out, n, iter, iter_block_infections);

		int* tmp = in;
		in = out;
		out = tmp;
	}

	// reduce infections per iter, which are stored per block
	threads_per_block = REDUCTION_BLOCK_SIZE;
	blocks_per_grid = iters;
	int blocks_per_iter = (grid_size.x * grid_size.y) / REDUCTION_BLOCK_SIZE;
	reduce_infections<<<blocks_per_grid, threads_per_block>>>(infections, iter_block_infections, iters, blocks_per_iter, grid_size);

	if (in != city) {
		hipMemcpy(city, in, size, hipMemcpyDeviceToDevice);
		hipFree(in);
	} else {
		hipFree(out);
	}

	hipFreeArray(tex_array_contacts);
	hipFree(iter_block_infections);
}
