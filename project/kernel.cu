
#include <hip/hip_runtime.h>
#define BLOCK_SIZE 32
#define REDUCTION_BLOCK_SIZE 1024

__global__ void make_iteration(const int* const contacts, const int* const in, const int n, const int iter, int* const out, int* const iter_block_infections)
{
	__shared__ int shared_iter_block_infections[BLOCK_SIZE][BLOCK_SIZE];
	shared_iter_block_infections[threadIdx.y][threadIdx.x] = 0;

	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int x = blockIdx.x * blockDim.x + threadIdx.x;

	if (y >= n || x >= n)
		return;

	int idx = y * n + x;

	int house_in = in[idx];
	int house_out;

	if (house_in > 0) { // infected
		house_out = --house_in == 0 ? -30 : house_in;
	} else if (house_in < 0) { // recovering, immune
		house_out = ++house_in;
	} else { // healthy
		// check neighbours
		int inf_neighbours = 0;
		for (int dy = -1; dy <= 1; ++dy)
		for (int dx = -1; dx <= 1; ++dx) {
			// check bounds
			if ((x + dx < 0) || (x + dx >= n) || (y + dy < 0) || (y + dy >= n) || (dx == 0 && dy == 0))
				continue;

			inf_neighbours += (in[(y + dy) * n + (x + dx)] > 0) ? 1 : 0;
		}

		// compare to connectivity
		if (inf_neighbours > contacts[idx]) {
			house_out = 10;
			++shared_iter_block_infections[threadIdx.y][threadIdx.x];
		} else {
			house_out = 0;
		}
	}
	out[idx] = house_out;

	__syncthreads();

	// reduction
	// sum and save the total number of new infections in this iteration per block

	// BLOCK_SIZE / 2 x BLOCK_SIZE / 2
	int reduction_block_size = BLOCK_SIZE / 2;
	if ((threadIdx.x % reduction_block_size == 0) && (threadIdx.y % reduction_block_size == 0)) {
		for (int yy = 0; yy < reduction_block_size; ++yy) {
		for (int xx = 0; xx < reduction_block_size; ++xx) {
			if (xx + yy == 0)
				continue;
			shared_iter_block_infections[threadIdx.y][threadIdx.x] += 
				shared_iter_block_infections[threadIdx.y + yy][threadIdx.x + xx];
		}
		}
	}

	__syncthreads();

	if (threadIdx.x == 0 && threadIdx.y == 0) {
		int iter_block_idx = (iter * gridDim.x * gridDim.y) + (blockIdx.y * gridDim.x + blockIdx.x);
		iter_block_infections[iter_block_idx] = 0;
		for (int yy = 0; yy < BLOCK_SIZE; yy += reduction_block_size)
		for (int xx = 0; xx < BLOCK_SIZE; xx += reduction_block_size)
			iter_block_infections[iter_block_idx] += shared_iter_block_infections[yy][xx];
	}
}

/*
   For each iteration, sums infections per block to compute the number of new infections per iteration.
*/
__global__ void reduce_infections(int* const infections, const int* const iter_block_infections, const int iters, const int blocks_per_iter, const int grid_size)
{
	__shared__ int shared[REDUCTION_BLOCK_SIZE];

	const int tid = threadIdx.x;
	const int iter = blockIdx.x;
	const int infections_idx = iter * grid_size * grid_size;
	shared[tid] = iter_block_infections[infections_idx + tid];

	// 2048 / 1024
	for (int i = 1; i < blocks_per_iter; ++i) {
		shared[tid] += iter_block_infections[infections_idx + tid + (i * blockDim.x)];
	}

	__syncthreads();

	for (unsigned int s = 1; s < blockDim.x; s *= 2) {
		if (tid % (2 * s) == 0) {
			shared[tid] += shared[tid + s];
		}
		__syncthreads();
	}

	if (tid == 0) {
		infections[iter] = shared[0];
	}
}

void solveGPU(const int* const contacts, int* const city, int* const infections, const int n, const int iters)
{
	int grid_size = ceil(n / (float) BLOCK_SIZE);

	int *in = city;
	int *out;
	int *iter_block_infections; 	// [iters][grid_size][grid_size] 
					// 3D array storing infections per block per iteration

	if ((hipMalloc((void**)&out, n * n * sizeof(int)) != hipSuccess)
			|| (hipMalloc((void**)&iter_block_infections, iters * grid_size * grid_size * sizeof(int)) != hipSuccess)) {
		fprintf(stderr, "hipMalloc failed\n");
		return;
	}

	dim3 threads_per_block = dim3(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocks_per_grid = dim3(grid_size, grid_size);
	for (int iter = 0; iter < iters; ++iter) {
		make_iteration<<<blocks_per_grid, threads_per_block>>>(contacts, in, n, iter, out, iter_block_infections);
		int *tmp = in;
		in = out;
		out = tmp;
	}

	// reduce infections per iter, which are stored per block
	threads_per_block = REDUCTION_BLOCK_SIZE;
	blocks_per_grid = iters;
	int blocks_per_iter = (grid_size * grid_size) / REDUCTION_BLOCK_SIZE;
	reduce_infections<<<blocks_per_grid, threads_per_block>>>(infections, iter_block_infections, iters, blocks_per_iter, grid_size);

	if (in != city) {
		hipMemcpy(city, in, n*n*sizeof(int), hipMemcpyDeviceToDevice);
		hipFree(in);
	} else {
		hipFree(out);
	}

	hipFree(iter_block_infections);
}
