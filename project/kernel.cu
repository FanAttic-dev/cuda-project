
#include <hip/hip_runtime.h>
// write your code into this file
// your kernels can be implemented directly here, or included
// function solveGPU is a device function: it can allocate memory, call CUDA kernels etc.

#define BLOCK_SIZE 16

__global__ void make_iteration(const int* const contacts, const int* const in, const int n, const int iter, int* const out, int* const iter_block_infections)
{
	__shared__ int shared_iter_block_infections[BLOCK_SIZE][BLOCK_SIZE];
	shared_iter_block_infections[threadIdx.y][threadIdx.x] = 0;

	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	int idx = i * n + j;

	int house_in = in[idx];
	int house_out = -1;

	if (house_in > 0) {
		// infected
		house_out = --house_in == 0 ? -30 : house_in;
	} else if (house_in < 0) {
		// recovering, immune
		house_out = ++house_in;
	} else {
		// healthy

		// check neighbours
		int inf_neighbours = 0;
		for (int ii = max(0, i-1); ii <= min(i+1, n-1); ++ii)
			for (int jj = max(0, j-1); jj <= min(j+1, n-1); ++jj)
				if (in[ii * n + jj] > 0)
					++inf_neighbours;

		// compare to connectivity
		if (inf_neighbours > contacts[idx]) {
			house_out = 10;
			shared_iter_block_infections[threadIdx.y][threadIdx.x] = 1;
		} else {
			house_out = 0;
		}
	}
	out[idx] = house_out;

	__syncthreads();

	if (threadIdx.x == 0 && threadIdx.y == 0) {
		iter_block_infections[blockIdx.y * gridDim.x + blockIdx.x] = 0;
		for (int ii = 0; ii < BLOCK_SIZE; ++ii) {
			for (int jj = 0; jj < BLOCK_SIZE; ++jj) {
				iter_block_infections[(iter * gridDim.x * gridDim.y) + (blockIdx.y * gridDim.x + blockIdx.x)] += shared_iter_block_infections[ii][jj];
			}
		}
	}
}

__global__ void sum_infections_per_iteration(int* const infections, const int* const iter_block_infections, const int iter, const int grid_size)
{
	if (threadIdx.x != 0)
		return;

	infections[iter] = 0;

	for (int ii = 0; ii < grid_size; ++ii) {
		for (int jj = 0; jj < grid_size; ++jj) {
			infections[iter] += iter_block_infections[(iter * grid_size * grid_size) + (ii * grid_size + jj)];
		}
	}
}

void solveGPU(const int* const contacts, int* const city, int* const infections, const int n, const int iters)
{
	int *in = city;
	int *out;
	int *iter_block_infections; // 3D array storing infections per block per iteration

	int grid_size = ceil(n/BLOCK_SIZE);
	printf("Grid count: %d\n", grid_size);

	if (hipMalloc((void**)&out, n*n*sizeof(int)) != hipSuccess
			|| hipMalloc((void**)&iter_block_infections, iters * grid_size * grid_size * sizeof(int)) != hipSuccess) {
		fprintf(stderr, "hipMalloc failed\n");
		return;
	}

	dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocksPerGrid(grid_size, grid_size);

	for (int iter = 0; iter < iters; ++iter) {
		make_iteration<<<blocksPerGrid, threadsPerBlock>>>(contacts, in, n, iter, out, iter_block_infections);

		sum_infections_per_iteration<<<1, 1>>>(infections, iter_block_infections, iter, grid_size);

		int *tmp = in;
		in = out;
		out = tmp;
	}

	if (in != city) {
		hipMemcpy(city, in, n*n*sizeof(int), hipMemcpyDeviceToDevice);
		hipFree(in);
	} else {
		hipFree(out);
	}

	hipFree(iter_block_infections);
}
