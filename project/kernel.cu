
#include <hip/hip_runtime.h>
#define BLOCK_SIZE 16 // max 32
#define REDUCTION_BLOCK_SIZE 512

__device__ void warp_reduce(volatile int *shared_data, int tid)
{
	shared_data[tid] += shared_data[tid + 32];
	shared_data[tid] += shared_data[tid + 16];
	shared_data[tid] += shared_data[tid + 8];
	shared_data[tid] += shared_data[tid + 4];
	shared_data[tid] += shared_data[tid + 2];
	shared_data[tid] += shared_data[tid + 1];
}

__device__ bool check_neighbours_global(const int* const in, const int n, const int x, const int y, const int threshold)
{	
	// check neighbours
	int inf_neighbours = 0;
	for (int dx = -1; dx <= 1; ++dx) 
	for (int dy = -1; dy <= 1; ++dy) {
		// check bounds
		if ((dx == 0 && dy == 0) || (x + dx < 0) || (x + dx >= n) || (y + dy < 0) || (y + dy >= n))
			continue;

		if (in[(y + dy) * n + (x + dx)] > 0)
			++inf_neighbours;
	}

	return inf_neighbours > threshold;
}

__global__ void make_iteration(const int* const contacts, const int* const in, const int n, const int iter, int* const out, int* const iter_block_infections)
{
	__shared__ int shared_iter_block_infections[BLOCK_SIZE * BLOCK_SIZE];
	const int idx_local = threadIdx.y * blockDim.x + threadIdx.x;
	shared_iter_block_infections[idx_local] = 0;

	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= n || y >= n)
		return;

	const int idx_global = y * n + x;
	const int house_in = in[idx_global];
	int house_out = 0;

	if (house_in == 0) { // healthy
		const int threshold = contacts[idx_global];
		bool infected = check_neighbours_global(in, n, x, y, threshold);

		// compare to connectivity
		if (infected) {
			house_out = 10;
			++shared_iter_block_infections[idx_local];
		}
	} else if (house_in > 0) { // infected
		house_out = house_in - 1 == 0 ? -30 : house_in - 1;
	} else { // (house_in < 0) // recovering, immune
		house_out = house_in + 1;
	} 
	__syncthreads();

/*	
	// sum and save the total number of new infections in this iteration per block
	if (idx_local == 0) {
		int iter_block_idx = (iter * gridDim.x * gridDim.y) + (blockIdx.y * gridDim.x + blockIdx.x);
		iter_block_infections[iter_block_idx] = 0;
		for (int yy = 0; yy < BLOCK_SIZE; ++yy)
		for (int xx = 0; xx < BLOCK_SIZE; ++xx)
			iter_block_infections[iter_block_idx] += shared_iter_block_infections[yy * BLOCK_SIZE + xx];
	}
*/

	// reduction
	// sum and save the total number of new infections in this iteration per block
	for (unsigned int s = BLOCK_SIZE * BLOCK_SIZE / 2; s > 32; s >>= 1) {
		if (idx_local < s) {
			shared_iter_block_infections[idx_local] += shared_iter_block_infections[idx_local + s];
		}
		__syncthreads();
	}
/*
	// TODO make generic
	if (idx_local < 128) {
		shared_iter_block_infections[idx_local] += shared_iter_block_infections[idx_local + 128];
		__syncthreads();
	}

	if (idx_local < 64) {
		shared_iter_block_infections[idx_local] += shared_iter_block_infections[idx_local + 64];
		__syncthreads();
	}
*/

	if (idx_local < 32) {
		warp_reduce(shared_iter_block_infections, idx_local);
		__syncthreads();
	}

	if (idx_local == 0) {
		int iter_block_idx = (iter * gridDim.x * gridDim.y) + (blockIdx.y * gridDim.x + blockIdx.x);
		iter_block_infections[iter_block_idx] = shared_iter_block_infections[0];
	}

	out[idx_global] = house_out;
}

/*
   For each iteration, sums infections per block to compute the number of new infections per iteration.
*/
__global__ void reduce_infections(int* const infections, const int* const iter_block_infections, const int iters, const int blocks_per_iter, const int grid_size)
{
	__shared__ int shared[REDUCTION_BLOCK_SIZE];

	const int tid = threadIdx.x;
	const int iter = blockIdx.x;
	const int infections_idx = iter * grid_size * grid_size;
	shared[tid] = iter_block_infections[infections_idx + tid];

	for (int i = 1; i < blocks_per_iter; ++i) {
		shared[tid] += iter_block_infections[infections_idx + tid + (i * blockDim.x)];
	}

	__syncthreads();

	for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
		if (tid < s)
			shared[tid] += shared[tid + s];

		__syncthreads();
	}

	if (tid < 32) {
		warp_reduce(shared, tid);
		__syncthreads();
	}

	if (tid == 0) {
		infections[iter] = shared[0];
	}
}

void solveGPU(const int* const contacts, int* const city, int* const infections, const int n, const int iters)
{
	int grid_size = ceil(n / (float) BLOCK_SIZE);

	int *in = city;
	int *out;
	int *iter_block_infections; 	// [iters][grid_size][grid_size] 
					// 3D array storing infections per block per iteration

	if ((hipMalloc((void**)&out, n * n * sizeof(int)) != hipSuccess)
			|| (hipMalloc((void**)&iter_block_infections, iters * grid_size * grid_size * sizeof(int)) != hipSuccess)) {
		fprintf(stderr, "hipMalloc failed\n");
		return;
	}

	dim3 threads_per_block = dim3(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocks_per_grid = dim3(grid_size, grid_size);
	for (int iter = 0; iter < iters; ++iter) {
		make_iteration<<<blocks_per_grid, threads_per_block>>>(contacts, in, n, iter, out, iter_block_infections);
		int *tmp = in;
		in = out;
		out = tmp;
	}

	// reduce infections per iter, which are stored per block
	threads_per_block = REDUCTION_BLOCK_SIZE;
	blocks_per_grid = iters;
	int blocks_per_iter = (grid_size * grid_size) / REDUCTION_BLOCK_SIZE;
	reduce_infections<<<blocks_per_grid, threads_per_block>>>(infections, iter_block_infections, iters, blocks_per_iter, grid_size);

	if (in != city) {
		hipMemcpy(city, in, n*n*sizeof(int), hipMemcpyDeviceToDevice);
		hipFree(in);
	} else {
		hipFree(out);
	}

	hipFree(iter_block_infections);
}
