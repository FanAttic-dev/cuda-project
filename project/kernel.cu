
#include <hip/hip_runtime.h>
// write your code into this file
// your kernels can be implemented directly here, or included
// function solveGPU is a device function: it can allocate memory, call CUDA kernels etc.

#define BLOCK_SIZE 8

__global__ void make_iteration(const int* const contacts, const int* const in, const int n, const int iter, int* const out, int* const iter_block_infections)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	int idx = i * n + j;

	if (idx >= n * n)
		return;

	__shared__ int shared_iter_block_infections[BLOCK_SIZE][BLOCK_SIZE];
	shared_iter_block_infections[threadIdx.y][threadIdx.x] = 0;

	int house_in = in[idx];
	int house_out = -1;

	if (house_in > 0) {
		// infected
		house_out = --house_in == 0 ? -30 : house_in;
	} else if (house_in < 0) {
		// recovering, immune
		house_out = ++house_in;
	} else {
		// healthy

		// check neighbours
		int inf_neighbours = 0;
		for (int ii = max(0, i-1); ii <= min(i+1, n-1); ++ii)
		for (int jj = max(0, j-1); jj <= min(j+1, n-1); ++jj)
			if (in[ii * n + jj] > 0)
				++inf_neighbours;

		// compare to connectivity
		if (inf_neighbours > contacts[idx]) {
			house_out = 10;
			++shared_iter_block_infections[threadIdx.y][threadIdx.x];
		} else {
			house_out = 0;
		}
	}
	out[idx] = house_out;

	__syncthreads();

	if (threadIdx.x == 0 && threadIdx.y == 0) {
		iter_block_infections[(iter * gridDim.x * gridDim.y) + (blockIdx.y * gridDim.x + blockIdx.x)] = 0;
		for (int ii = 0; ii < BLOCK_SIZE; ++ii)
		for (int jj = 0; jj < BLOCK_SIZE; ++jj) {
			iter_block_infections[(iter * gridDim.x * gridDim.y) + (blockIdx.y * gridDim.x + blockIdx.x)] += shared_iter_block_infections[ii][jj];
		}
	}
}

__global__ void reduce_infections(int* const infections, const int* const iter_block_infections, const int iters, const int grid_size)
{
	int iter = blockIdx.x * blockDim.x + threadIdx.x;

	if (iter >= iters)
		return;

	infections[iter] = 0;

	for (int ii = 0; ii < grid_size; ++ii)
	for (int jj = 0; jj < grid_size; ++jj)
		infections[iter] += iter_block_infections[(iter * grid_size * grid_size) + (ii * grid_size + jj)];
}

void solveGPU(const int* const contacts, int* const city, int* const infections, const int n, const int iters)
{
	int *in = city;
	int *out;
	int *iter_block_infections; // 3D array storing infections per block per iteration

	int grid_size = ceil(n / (float) BLOCK_SIZE);
	printf("Block count: %d\n", grid_size);

	if (hipMalloc((void**)&out, n*n*sizeof(int)) != hipSuccess
			|| hipMalloc((void**)&iter_block_infections, iters * grid_size * grid_size * sizeof(int)) != hipSuccess) {
		fprintf(stderr, "hipMalloc failed\n");
		return;
	}

	dim3 threads_per_block = dim3(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocks_per_grid = dim3(grid_size, grid_size);
	for (int iter = 0; iter < iters; ++iter) {
		make_iteration<<<blocks_per_grid, threads_per_block>>>(contacts, in, n, iter, out, iter_block_infections);

		int *tmp = in;
		in = out;
		out = tmp;
	}

	threads_per_block = 32;
	blocks_per_grid = ceil(iters / (float) threads_per_block.x);
	reduce_infections<<<blocks_per_grid, threads_per_block>>>(infections, iter_block_infections, iters, grid_size);

	if (in != city) {
		hipMemcpy(city, in, n*n*sizeof(int), hipMemcpyDeviceToDevice);
		hipFree(in);
	} else {
		hipFree(out);
	}

	hipFree(iter_block_infections);
}
