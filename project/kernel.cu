
#include <hip/hip_runtime.h>
#define BLOCK_SIZE 32
#define REDUCTION_BLOCK_SIZE 1024

__global__ void make_iteration(const int* const contacts, const int* const in, const int n, const int iter, int* const out, int* const iter_block_infections)
{
	__shared__ int shared_iter_block_infections[BLOCK_SIZE * BLOCK_SIZE];
	int idx_local = threadIdx.y * blockDim.x + threadIdx.x;
	shared_iter_block_infections[idx_local] = 0;

	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int x = blockIdx.x * blockDim.x + threadIdx.x;

	if (y >= n || x >= n)
		return;

	int idx_global = y * n + x;

	int house_in = in[idx_global];
	int house_out;

	if (house_in > 0) { // infected
		house_out = --house_in == 0 ? -30 : house_in;
	} else if (house_in < 0) { // recovering, immune
		house_out = ++house_in;
	} else { // healthy
		// check neighbours
		int inf_neighbours = 0;
		for (int dy = -1; dy <= 1; ++dy)
		for (int dx = -1; dx <= 1; ++dx) {
			// check bounds
			if ((x + dx < 0) || (x + dx >= n) || (y + dy < 0) || (y + dy >= n) || (dx == 0 && dy == 0))
				continue;

			inf_neighbours += (in[(y + dy) * n + (x + dx)] > 0) ? 1 : 0;
		}

		// compare to connectivity
		if (inf_neighbours > contacts[idx_global]) {
			house_out = 10;
			++shared_iter_block_infections[idx_local];
		} else {
			house_out = 0;
		}
	}
	out[idx_global] = house_out;

	__syncthreads();

	// reduction
	// sum and save the total number of new infections in this iteration per block
	for (unsigned int s = (BLOCK_SIZE * BLOCK_SIZE) / 2; s > 0; s >>= 1) {
		if (idx_local < s) {	
			shared_iter_block_infections[idx_local] += shared_iter_block_infections[idx_local + s];
		}
		__syncthreads();
	}

	if (threadIdx.x == 0 && threadIdx.y == 0) {
		int iter_block_idx = (iter * gridDim.x * gridDim.y) + (blockIdx.y * gridDim.x + blockIdx.x);
		iter_block_infections[iter_block_idx] = shared_iter_block_infections[0];
	}
}

/*
   For each iteration, sums infections per block to compute the number of new infections per iteration.
*/
__global__ void reduce_infections(int* const infections, const int* const iter_block_infections, const int iters, const int blocks_per_iter, const int grid_size)
{
	__shared__ int shared[REDUCTION_BLOCK_SIZE];

	const int tid = threadIdx.x;
	const int iter = blockIdx.x;
	const int infections_idx = iter * grid_size * grid_size;
	shared[tid] = iter_block_infections[infections_idx + tid];

	// 2048 / 1024 = 2
	for (int i = 1; i < blocks_per_iter; ++i) {
		shared[tid] += iter_block_infections[infections_idx + tid + (i * blockDim.x)];
	}

	__syncthreads();

	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
		if (tid < s)
			shared[tid] += shared[tid + s];

		__syncthreads();
	}

	// TODO unroll

	if (tid == 0) {
		infections[iter] = shared[0];
	}
}

void solveGPU(const int* const contacts, int* const city, int* const infections, const int n, const int iters)
{
	int grid_size = ceil(n / (float) BLOCK_SIZE);

	int *in = city;
	int *out;
	int *iter_block_infections; 	// [iters][grid_size][grid_size] 
					// 3D array storing infections per block per iteration

	if ((hipMalloc((void**)&out, n * n * sizeof(int)) != hipSuccess)
			|| (hipMalloc((void**)&iter_block_infections, iters * grid_size * grid_size * sizeof(int)) != hipSuccess)) {
		fprintf(stderr, "hipMalloc failed\n");
		return;
	}

	dim3 threads_per_block = dim3(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocks_per_grid = dim3(grid_size, grid_size);
	for (int iter = 0; iter < iters; ++iter) {
		make_iteration<<<blocks_per_grid, threads_per_block>>>(contacts, in, n, iter, out, iter_block_infections);
		int *tmp = in;
		in = out;
		out = tmp;
	}

	// reduce infections per iter, which are stored per block
	threads_per_block = REDUCTION_BLOCK_SIZE;
	blocks_per_grid = iters;
	int blocks_per_iter = (grid_size * grid_size) / REDUCTION_BLOCK_SIZE;
	reduce_infections<<<blocks_per_grid, threads_per_block>>>(infections, iter_block_infections, iters, blocks_per_iter, grid_size);

	if (in != city) {
		hipMemcpy(city, in, n*n*sizeof(int), hipMemcpyDeviceToDevice);
		hipFree(in);
	} else {
		hipFree(out);
	}

	hipFree(iter_block_infections);
}
