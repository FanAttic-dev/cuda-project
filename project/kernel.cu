
#include <hip/hip_runtime.h>
#define BLOCK_SIZE_X 32
#define BLOCK_SIZE_Y 16
#define REDUCTION_BLOCK_SIZE 32

__inline__ __device__ bool check_neighbours_global_border(const int* const in, const int n, const int x, const int y, const int threshold)
{	
	// check neighbours
	int infected_neighbours = 0;
	for (int dy = -1; dy <= 1; ++dy) {
		for (int dx = -1; dx <= 1; ++dx) {
			// check bounds
			if ((dx == 0 && dy == 0) || (x + dx < 0) || (x + dx >= n) || (y + dy < 0) || (y + dy >= n))
				continue;

			if (in[(y + dy) * n + (x + dx)] > 0)
				++infected_neighbours;
		}
	}

	return infected_neighbours > threshold;
}

__inline__ __device__ bool check_neighbours_global_inner(const int* const in, const int n, const int x, const int y, const int threshold)
{	
	// check neighbours
	int infected_neighbours = 0;

	for (int dy = -1; dy <= 1; ++dy) {
		int row = (y + dy) * n;
		infected_neighbours += in[row + (x - 1)] > 0 ? 1 : 0;
		infected_neighbours += in[row + x] > 0 ? 1 : 0;
		infected_neighbours += in[row + (x + 1)] > 0 ? 1 : 0;
	}

	return infected_neighbours > threshold;
}

__global__ void make_iteration(const int* const contacts, const int* const in, const int n, const int iter, int* const out, int* const iter_block_infections)
{
	__shared__ int shared_iter_block_infections[BLOCK_SIZE_X * BLOCK_SIZE_Y];
	const int idx_local = threadIdx.y * blockDim.x + threadIdx.x;
	shared_iter_block_infections[idx_local] = 0;

	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= n || y >= n)
		return;

	const int idx_global = y * n + x;
	const int house_in = in[idx_global];
	int house_out = 0;

	if (house_in == 0) { // healthy
		bool infected;	
		if (x == 0 || y == 0 || x == n-1 || y == n-1)
			infected = check_neighbours_global_border(in, n, x, y, contacts[idx_global]);
		else
			infected = check_neighbours_global_inner(in, n, x, y, contacts[idx_global]);

		if (infected) {
			house_out = 10;
			++shared_iter_block_infections[idx_local];
		}
	} else if (house_in > 0) { // infected
		house_out = house_in - 1 == 0 ? -30 : house_in - 1;
	} else { // (house_in < 0) // recovering, immune
		house_out = house_in + 1;
	}

	__syncthreads();

	// reduction

	// sum and save the total number of new infections in this iteration per block
	int mySum = shared_iter_block_infections[idx_local];
	for (unsigned int offset = (BLOCK_SIZE_X * BLOCK_SIZE_Y) >> 1; offset > 32; offset >>= 1) {
		if (idx_local < offset)
			shared_iter_block_infections[idx_local] = mySum = mySum + shared_iter_block_infections[idx_local + offset];

		__syncthreads();
	}

	// reduce last warp
	if (idx_local < 32) {
		mySum += shared_iter_block_infections[idx_local + 32];

		for (int offset = warpSize >> 1; offset > 0; offset >>= 1)
			mySum += __shfl_down_sync(0xffffffff, mySum, offset);
	}

	if (idx_local == 0) {
		const unsigned int iter_block_idx = (iter * gridDim.x * gridDim.y) + (blockIdx.y * gridDim.x + blockIdx.x);
		iter_block_infections[iter_block_idx] = mySum;
	}

	out[idx_global] = house_out;
}

/*
   For each iteration, sums infections per block to compute the number of new infections per iteration.
*/
__global__ void reduce_infections(int* const infections, const int* const iter_block_infections, const int iters, const int blocks_per_iter, const dim3 grid_size)
{
	__shared__ int shared[REDUCTION_BLOCK_SIZE];

	const int tid = threadIdx.x;
	const int iter = blockIdx.x;
	const int infections_idx = iter * grid_size.x * grid_size.y; //+ tid;

	int mySum = 0;
	if (tid == 0) {
		for (int i = 0; i < grid_size.x * grid_size.y; ++i)
			mySum += iter_block_infections[infections_idx + i];
	}

/*
	int mySum = iter_block_infections[infections_idx];
	for (int i = 1; i < blocks_per_iter; ++i) {
		shared[tid] = mySum = mySum + iter_block_infections[infections_idx + (i * REDUCTION_BLOCK_SIZE)];
	}

	__syncthreads();


	for (unsigned int offset = blockDim.x >> 1; offset > 32; offset >>= 1) {
		if (tid < offset)
			shared[tid] = mySum = mySum + shared[tid + offset];

		__syncthreads();
	}

	if (tid < 32) {
		mySum += shared[tid + 32];

		for (int offset = warpSize >> 1; offset > 0; offset >>= 1)
			mySum += __shfl_down_sync(0xffffffff, mySum, offset);
	}
*/
	if (tid == 0) {
		infections[iter] = mySum;
	}
}

void solveGPU(const int* const contacts, int* const city, int* const infections, const int n, const int iters)
{
	dim3 grid_size = dim3(ceil(n / (float) BLOCK_SIZE_X), ceil(n / (float) BLOCK_SIZE_Y));

	int *in = city;
	int *out;
	int *iter_block_infections; 	// [iters][grid_size][grid_size] 
					// 3D array storing infections per block per iteration

	if ((hipMalloc((void**)&out, n * n * sizeof(int)) != hipSuccess)
			|| (hipMalloc((void**)&iter_block_infections, iters * grid_size.x * grid_size.y * sizeof(int)) != hipSuccess)) {
		fprintf(stderr, "hipMalloc failed\n");
		return;
	}

	dim3 threads_per_block = dim3(BLOCK_SIZE_X, BLOCK_SIZE_Y);
	dim3 blocks_per_grid = dim3(grid_size.x, grid_size.y);
	for (int iter = 0; iter < iters; ++iter) {
		make_iteration<<<blocks_per_grid, threads_per_block>>>(contacts, in, n, iter, out, iter_block_infections);

		int *tmp = in;
		in = out;
		out = tmp;
	}

	//printf("Grid size: %d x %d\n", grid_size, grid_size);

	// reduce infections per iteration
	threads_per_block = REDUCTION_BLOCK_SIZE;
	blocks_per_grid = iters;
	int blocks_per_iter = ceil((grid_size.x * grid_size.y) / REDUCTION_BLOCK_SIZE);
	reduce_infections<<<blocks_per_grid, threads_per_block>>>(infections, iter_block_infections, iters, blocks_per_iter, grid_size);

	if (in != city) {
		hipMemcpy(city, in, n*n*sizeof(int), hipMemcpyDeviceToDevice);
		hipFree(in);
	} else {
		hipFree(out);
	}

	hipFree(iter_block_infections);
}
